#include "hip/hip_runtime.h"
#include "cost_function_kernel.h"
#include "soft_constraints.h"

#include "../gpu_displacement_field.h"

#include <stk/math/float4.h>

namespace cuda {
    using namespace stk::cuda;
}

template<typename T>
struct SoftConstraintsImpl
{
    typedef T VoxelType;

    SoftConstraintsImpl(
        const cuda::VolumePtr<uint8_t>& constraint_mask,
        const cuda::VolumePtr<float4>& constraint_values
    ) : _constraint_mask(constraint_mask), _constraint_values(constraint_values)
    {}

    __device__ float operator()(
        const cuda::VolumePtr<VoxelType>& fixed,
        const cuda::VolumePtr<VoxelType>& moving,
        const dim3& /*fixed_dims*/,
        const dim3& /*moving_dims*/,
        const int3& fixed_p,
        const float3& /*moving_p*/,
        const float3& d
    )
    {
        float4 diff = float4{d.x, d.y, d.z, 0.0f} - _constraint_values(fixed_p.x, fixed_p.y, fixed_p.z);
        float dist2 = fminf(stk::norm2(diff), 100000.0f); // Clamp to avoid explosion
        return float(_constraint_mask(fixed_p.x, fixed_p.y, fixed_p.z)) * dist2;
    }

    cuda::VolumePtr<uint8_t> _constraint_mask;
    cuda::VolumePtr<float4> _constraint_values;
};

void GpuCostFunction_SoftConstraints::cost(
    GpuDisplacementField& df,
    const float3& delta,
    float weight,
    const int3& offset,
    const int3& dims,
    Settings::UpdateRule update_rule,
    stk::GpuVolume& cost_acc,
    stk::cuda::Stream& stream
)
{
    ASSERT(cost_acc.voxel_type() == stk::Type_Float2);

    FATAL_IF(_constraint_mask.voxel_type() != stk::Type_UChar ||
             _constraint_values.voxel_type() != stk::Type_Float4)
        << "Unsupported pixel type";

    auto kernel = CostFunctionKernel<SoftConstraintsImpl<float>>(
        SoftConstraintsImpl<float>(
            _constraint_mask,
            _constraint_values
        ),
        // This cost function isn't using either fixed nor moving volume
        stk::GpuVolume(),
        stk::GpuVolume(),
        _fixed_mask,
        _moving_mask,
        weight,
        cost_acc
    );

    invoke_cost_function_kernel(
        kernel,
        delta,
        offset,
        dims,
        df,
        update_rule,
        stream
    );
}

